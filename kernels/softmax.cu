#include "hip/hip_runtime.h"
#include "util.cu"
#ifdef __HIP_PLATFORM_AMD__
#include <hip/hip_runtime.h>
#endif

// it is assumed that we will only be using this on matrixs with small number of columns
// (so the perf won't be terrible)
__global__ void softmaxAcrossColumnsKernel(const size_t rows, const size_t cols, const float* input, float* output)
{
    const size_t tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= cols)
        return;

    const float* thisColumn = input + rows * tid;
    float* thisOutput = output + rows * tid;

    float maximum = -100000.0F;

    for (size_t i = 0; i < rows; i++) {
        maximum = max(maximum, thisColumn[i]);
    }

    float total = 0.0F;

    for (size_t i = 0; i < rows; i++) {
        const float exp = expf(thisColumn[i] - maximum);
        thisOutput[i] = exp;
        total += exp;
    }

    for (size_t i = 0; i < rows; i++) {
        thisOutput[i] /= total;
    }
}

__global__ void crossEntropyKernel(const size_t size, const float* pred, const float* target, float* out)
{
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= size)
        return;

    out[i] = (target[i] == 0.0F) ? 0.0F : -target[i] * logf(pred[i]);
}

__global__ void backpropSoftmaxCrossEntropyKernel(
    const size_t size,
    const float* softmaxed,
    const float* target,
    const float* out_grad,
    float* input_grad)
{
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= size)
        return;

    input_grad[i] += (softmaxed[i] - target[i]) * out_grad[0];
}

extern "C" void softmax_across_columns(const size_t rows, const size_t cols, const float* input, float* output)
{
    const size_t grid_x = (cols + threadsPerBlock - 1) / threadsPerBlock;
    softmaxAcrossColumnsKernel<<<grid_x, threadsPerBlock>>>(rows, cols, input, output);
}

extern "C" void crossentropy(const size_t size, const float* pred, const float* target, float* out)
{
    const size_t numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    crossEntropyKernel<<<numBlocks, threadsPerBlock>>>(size, pred, target, out);
}

extern "C" void backprop_softmax_cross_entropy(
    const size_t size,
    const float* softmaxed,
    const float* target,
    const float* out_grad,
    float* input_grad)
{
    const size_t numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    backpropSoftmaxCrossEntropyKernel<<<numBlocks, threadsPerBlock>>>(size, softmaxed, target, out_grad, input_grad);
}
